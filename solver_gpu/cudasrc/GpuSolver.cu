#include "hip/hip_runtime.h"
#include "GpuSolverKernels.cuh"
#include "GpuSolver.h"
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <sys/time.h>
#include <time.h>
#include "rand_gpu.h"
#include "parameters.h"
//----------------------------------------------------------------------------------------------------------------------

GpuSolver::GpuSolver()
{

}

//----------------------------------------------------------------------------------------------------------------------

GpuSolver::~GpuSolver()
{
  if ( m_active )
  {
    hipFree( m_pvx );
    hipFree( m_pvy );
    hipFree( m_density );
    hipFree( m_pressure );
    hipFree( m_divergence );
    hipFree( m_velocity.x );
    hipFree( m_velocity.y );
    hipFree( m_previousVelocity.x );
    hipFree( m_previousVelocity.y );
    hipFree( m_previousDensity );

    free( m_cpuDensity );
    free( m_cpuPrevDensity );
    free( m_cpuPreviousVelocity.x );
    free( m_cpuPreviousVelocity.y );
  }
}

//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::setParameters()
{
  m_gridSize.x = Common::gridWidth;
  m_gridSize.y = Common::gridHeight;
  m_totCell = Common::totCells;
  m_rowVelocity.x = Common::rowVelocityX;
  m_rowVelocity.y = Common::rowVelocityY;

  m_columnVelocity.x = Common::columnVelocityX;
  m_columnVelocity.y = Common::columnVelocityY;

  m_totVelX = Common::totHorizontalVelocity;
  m_totVelY = Common::totVerticalVelocity;

  m_min.x = 0.0f;
  m_max.x = (real)m_gridSize.x;
  m_min.y = 0.0f;
  m_max.y = (real)m_gridSize.y;

  m_timeStep = 1.0f;
  m_diffusion = 0.0f;
  m_viscosity = 0.0f;
  m_inputDensity = 100.0;
}

//---------------------------------------------------------------------------------------------------------------------

void GpuSolver::allocateArrays()
{
  hipMalloc((void **) &m_pvx, sizeof(tuple<real>) * m_totVelX );
  hipMalloc((void **) &m_pvy, sizeof(tuple<real>) * m_totVelY );
  hipMalloc((void **) &m_density, sizeof(real)*m_totCell );
  hipMalloc((void **) &m_pressure, sizeof(real)*m_totCell );
  hipMalloc((void **) &m_divergence, sizeof(real)*m_totCell );
  hipMalloc((void **) &m_velocity.x, sizeof(real)*m_totVelX );
  hipMalloc((void **) &m_velocity.y, sizeof(real)*m_totVelY );
  hipMalloc((void **) &m_previousVelocity.x, sizeof(real)*m_totVelX );
  hipMalloc((void **) &m_previousVelocity.y, sizeof(real)*m_totVelY );
  hipMalloc((void **) &m_previousDensity, sizeof(real) * m_totCell );

  hipError_t err = hipGetLastError();
  if ( err != hipSuccess ) printf("malloc Error: %s\n", hipGetErrorString(err));

  m_cpuDensity = (real *) calloc( m_totCell, sizeof( real ) );
  m_cpuPrevDensity = (real *) calloc( m_totCell, sizeof( real ) );
  m_cpuPreviousVelocity.x = (real *) calloc( m_totVelX, sizeof(real) );
  m_cpuPreviousVelocity.y = (real *) calloc( m_totVelY, sizeof(real) );


  unsigned int tmp_gridSize[] = { m_gridSize.x, m_gridSize.y };
  unsigned int tmp_rowVelocity[] = { m_rowVelocity.x, m_rowVelocity.y };
  unsigned int tmp_columnVelocity[] = { m_columnVelocity.x, m_columnVelocity.y };
  int tmp_totVelocity[] = { m_totVelX, m_totVelY };

  hipMemcpyToSymbolAsync(HIP_SYMBOL(c_gridSize), tmp_gridSize, sizeof(unsigned int)*2 );
  hipMemcpyToSymbolAsync(HIP_SYMBOL(c_rowVelocity), tmp_rowVelocity, sizeof(int)*2,  0, hipMemcpyHostToDevice );
  hipMemcpyToSymbolAsync(HIP_SYMBOL(c_columnVelocity), tmp_columnVelocity, sizeof(int)*2,  0, hipMemcpyHostToDevice );
  hipMemcpyToSymbolAsync(HIP_SYMBOL(c_totVelocity), tmp_totVelocity, sizeof(int)*2,  0, hipMemcpyHostToDevice );

  err = hipGetLastError();
  if ( err != hipSuccess ) printf("copy Error during activation: %s\n", hipGetErrorString(err));
//  hipDeviceSynchronize();
//  std::cout << "memory allocated \n";
}

//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::activate()
{
  m_active = true;
  setParameters();
  allocateArrays();

  // 1024 -> max threads per block, in this case it will fire 16 blocks
  int nBlocks = m_totVelX / 1024;
  int blockDim = 1024 / m_gridSize.x + 1; // 9 threads per block

  dim3 block(blockDim, blockDim); // block of (X,Y) threads
  dim3 grid(nBlocks, nBlocks); // grid 2x2 blocks

  std::vector<hipStream_t> streams;
  streams.resize(2);

  for (auto &i : streams)
  {
    hipStreamCreate( &i );
  }

  d_setPvx<<<grid, block, 0, streams[0]>>>( m_pvx );
  d_setPvy<<<grid, block, 0, streams[1]>>>( m_pvy );

  hipError_t err = hipGetLastError();
  if ( err != hipSuccess ) printf("activation Error: %s\n", hipGetErrorString(err));
  cleanBuffer();
  reset();
//  std::cout << "solver activated \n";
}

//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::reset()
{
  // will have to change this one,
  // the kenel launch overhead it too damn high
  int threads = 1024;
  unsigned int densityBlocks = m_totCell / threads + 1;
  unsigned int xVelocityBlocks = m_totVelX / threads + 1;
  unsigned int yVelocityBlocks = m_totVelY / threads + 1;

  std::vector<hipStream_t> streams;
  streams.resize(4);

  for (auto &i : streams)
  {
    hipStreamCreate( &i );
  }

  d_reset<<<densityBlocks, threads, 0, streams[0]>>>(m_density, m_totCell);
  d_reset<<<densityBlocks, threads, 0, streams[1]>>>(m_divergence, m_totCell);
  d_reset<<<densityBlocks, threads, 0, streams[2]>>>(m_pressure, m_totCell);
  d_reset<<<densityBlocks, threads, 0, streams[3]>>>(m_previousDensity, m_totCell);
  d_reset<<<densityBlocks, threads, 0, streams[0]>>>(m_previousVelocity.x, m_totVelX);
  d_reset<<<densityBlocks, threads, 0, streams[1]>>>(m_previousVelocity.y, m_totVelY);
  d_reset<<<xVelocityBlocks, threads, 0, streams[2]>>>(m_velocity.x, m_totVelX);
  d_reset<<<yVelocityBlocks, threads, 0, streams[3]>>>(m_velocity.y, m_totVelY);

  hipError_t err = hipGetLastError();
  if ( err != hipSuccess ) printf("reset Error: %s\n", hipGetErrorString(err));
}

//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::cleanBuffer()
{
  int threads = 1024;
  unsigned int densityBlocks = m_totCell / threads + 1;
  unsigned int xVelocityBlocks = m_totVelX / threads + 1;
  unsigned int yVelocityBlocks = m_totVelY / threads + 1;

  std::vector<hipStream_t> streams;
  streams.resize(3);
  for (auto &i : streams)
  {
    hipStreamCreate( &i );
  }
  d_reset<<<densityBlocks, threads, 0, streams[0]>>>(m_previousDensity, m_totCell);
  d_reset<<<xVelocityBlocks, threads, 0, streams[1]>>>(m_previousVelocity.x, m_totVelX);
  d_reset<<<yVelocityBlocks, threads, 0, streams[2]>>>(m_previousVelocity.y, m_totVelY);

  memset( (void *) m_cpuPrevDensity, 0, sizeof(real) * m_totCell );
  memset( (void *) m_cpuPreviousVelocity.x, 0, sizeof(real) * m_totVelX );
  memset( (void *) m_cpuPreviousVelocity.y, 0, sizeof(real) * m_totVelY );

  hipError_t err = hipGetLastError();
  if ( err != hipSuccess ) printf("clean buffer Error: %s\n", hipGetErrorString(err));
}


//----------------------------------------------------------------------------------------------------------------------

const real * GpuSolver::getDens()
{
  copy( m_density, m_cpuDensity, m_totCell );
//  hipDeviceSynchronize();

  return m_cpuDensity;
}

//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::setVelBoundary( int flag )
{
  if( flag == 1 )
  {
    int threads = 1024;
    unsigned int blocks = std::max( m_columnVelocity.x, m_rowVelocity.x ) / threads + 1;
    d_setVelBoundaryX<<< blocks, threads>>>( m_velocity.x );
  }

  else if( flag == 2 )
  {
    int threads = 1024;
    unsigned int blocks = std::max( m_columnVelocity.y, m_rowVelocity.y ) / threads + 1;
    d_setVelBoundaryY<<< blocks, threads>>>( m_velocity.y );
  }
}

//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::setCellBoundary(real * _value, tuple<unsigned int> & _size )
{
  int threads = 1024;
  unsigned int blocks = std::max( _size.x, _size.y ) / threads + 1;

  d_setCellBoundary<<< blocks, threads >>>( _value );
}

//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::projection()
{
  int nBlocks = m_totCell / 1024;
  int blockDim = 1024 / m_gridSize.x + 1; // 9 threads per block
  unsigned int bins = blockDim * blockDim * sizeof(real);

  dim3 block(blockDim, blockDim); // block of (X,Y) threads
  dim3 grid(nBlocks, nBlocks); // grid 2x2 blocks

  hipMemsetAsync( (void *) m_pressure, 0, sizeof(real) * m_totCell );

  d_divergenceStep<<<grid, block, bins>>>( m_pressure, m_divergence, m_velocity );

  setCellBoundary( m_divergence, m_gridSize );

  for( unsigned int k = 0; k < 20; k++ )
  {
    d_projection<<<grid, block, bins>>>( m_pressure, m_divergence );
    setCellBoundary( m_pressure, m_gridSize );
  }

  d_velocityStep<<<grid, block, bins>>>( m_pressure, m_velocity );

  setVelBoundary(1);
  setVelBoundary(2);

  hipError_t err = hipGetLastError();
  if ( err != hipSuccess ) printf("Projection Error: %s\n", hipGetErrorString(err));
}

//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::advectVelocity()
{
  int nBlocks = m_totCell / 1024;
  int blockDim = 1024 / m_gridSize.x + 1;
  unsigned int bins = blockDim * blockDim * sizeof(real);

  dim3 block(blockDim, blockDim);
  dim3 grid(nBlocks, nBlocks);
  d_advectVelocity<<<grid, block, bins>>>( m_previousVelocity, m_velocity, m_pvx, m_pvy, m_timeStep );
  setVelBoundary(1);
  setVelBoundary(2);
  hipError_t err = hipGetLastError();
  if ( err != hipSuccess ) printf("Advection Error: %s\n", hipGetErrorString(err));
}

//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::advectCell()
{
  int nBlocks = m_totCell / 1024;
  int blockDim = 1024 / m_gridSize.x + 1;

  dim3 block(blockDim, blockDim);
  dim3 grid(nBlocks, nBlocks);
  d_advectCell<<<grid, block>>>( m_density, m_previousDensity, m_velocity, m_timeStep );
  setCellBoundary( m_density, m_gridSize );

  hipError_t err = hipGetLastError();
  if ( err != hipSuccess ) printf("Advection Error: %s\n", hipGetErrorString(err));
}

//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::diffuseVelocity()
{
  int nBlocks = m_totCell / 1024;
  int blockDim = 1024 / m_gridSize.x + 1;
  unsigned int bins = blockDim * blockDim * sizeof(real);

  dim3 block(blockDim, blockDim);
  dim3 grid(nBlocks, nBlocks);

  hipMemsetAsync( (void *)m_velocity.x, 0, sizeof(real)*m_totVelX );
  hipMemsetAsync( (void *)m_velocity.y, 0, sizeof(real)*m_totVelY );

  d_diffuseVelocity<<<grid, block, bins>>>( m_previousVelocity, m_velocity, m_timeStep, m_diffusion );

  hipError_t err = hipGetLastError();
  if ( err != hipSuccess ) printf("Diffusion Error: %s\n", hipGetErrorString(err));
}

//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::diffuseCell()
{
  int nBlocks = m_totCell / 1024;
  int blockDim = 1024 / m_gridSize.x + 1;
  unsigned int bins = blockDim * blockDim * sizeof(real);

  dim3 block(blockDim, blockDim);
  dim3 grid(nBlocks, nBlocks);
  hipMemsetAsync( (void *)m_density, 0, sizeof(real)*m_totCell );
  d_diffuseCell<<<grid, block, bins>>>( m_previousDensity, m_density, m_timeStep, m_viscosity );

  hipError_t err = hipGetLastError();
  if ( err != hipSuccess ) printf("Diffusion Error: %s\n", hipGetErrorString(err));
}

//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::exportCSV( std::string _file, tuple<real> * _t, int _sizeX, int _sizeY )
{
  std::ofstream out;
  out.open( _file );
  out.clear();
  int totSize = _sizeX * _sizeY;
  tuple<real> * result = (tuple<real> *) malloc( sizeof( tuple<real> ) * totSize );
  if( hipMemcpyAsync( result, _t, totSize * sizeof(tuple<real>), hipMemcpyDeviceToHost) != hipSuccess )
    exit(0);

  for(int i = 0; i < _sizeX; ++i)
  {
    for(int j = 0; j < _sizeY; ++j)
    {
      int idx = j * _sizeX + i;
      out << "( " << result[idx].x << ", " << result[idx].y << " )" << "; ";
    }
    out << "\n";
  }
  free( result );
}

//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::animVel()
{
  projection();

  if(m_diffusion > 0.0f)
  {
    SWAP(m_previousVelocity.x, m_velocity.x);
    SWAP(m_previousVelocity.y, m_velocity.y);
    diffuseVelocity();
  }

  SWAP(m_previousVelocity.x, m_velocity.x);
  SWAP(m_previousVelocity.y, m_velocity.y);
  advectVelocity();

  projection();
}

//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::animDen()
{
  if(m_viscosity > 0.0f)
  {
    SWAP(m_previousDensity, m_density);
    diffuseCell();
  }

  SWAP(m_previousDensity, m_density);
  advectCell();
}

//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::copy( tuple<real> * _src, tuple<real> * _dst, int _size )
{
  if( hipMemcpy( _dst, _src, _size * sizeof(tuple<real>), hipMemcpyDeviceToHost) != hipSuccess )
  {
    std::cout << "copy failed\n";
    exit(0);
  }
}

//----------------------------------------------------------------------------------------------------------------------


void GpuSolver::copy( real * _src, real * _dst, int _size )
{
//  hipDeviceSynchronize();
  if( hipMemcpy( _dst, _src, _size * sizeof( real ), hipMemcpyDeviceToHost) != hipSuccess )
  {
    hipError_t err = hipGetLastError();
    if ( err != hipSuccess ) printf("copy Error: %s\n", hipGetErrorString(err));

    std::cout << _size << " size\n";

    std::cout << "copy failed\n";
    exit(0);
  }
//  hipDeviceSynchronize();

}

//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::copyToDevice( real * _src, real * _dst, int _size )
{
  if( hipMemcpy( _dst, _src, _size * sizeof( real ), hipMemcpyHostToDevice ) != hipSuccess )
  {
    std::cout << "copy to device failed\n";
    exit(0);
  }
  hipDeviceSynchronize();

}

//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::gather( real * _value, unsigned int _size )
{
  int threads = 1024;
  unsigned int blocks = _size / threads + 1;

  real * d_values;
  hipMalloc( &d_values, sizeof(real) * _size );
  if( hipMemcpyAsync( d_values, _value, _size * sizeof( real ), hipMemcpyHostToDevice) != hipSuccess )
    exit(0);

  unsigned int bins = 10;
  d_gather<<< blocks, threads, bins * sizeof(real)>>>( d_values, _size );

  copy( d_values, _value, _size );
}

//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::randomizeArrays()
{
  Rand_GPU::randFloats( m_pressure, m_totCell );
  Rand_GPU::randFloats( m_divergence, m_totCell );
  Rand_GPU::randFloats( m_velocity.x, m_totVelX );
  Rand_GPU::randFloats( m_velocity.y, m_totVelY );
}

//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::addSource()
{
  int threads = 1024;
  unsigned int densityBlocks = m_totCell / threads + 1;
  unsigned int xVelocityBlocks = m_totVelX / threads + 1;
  unsigned int yVelocityBlocks = m_totVelY / threads + 1;
  std::vector<hipStream_t> streams;
  streams.resize(3);

  copyToDevice( m_cpuPrevDensity, m_previousDensity, m_totCell );
  copyToDevice( m_cpuPreviousVelocity.x, m_previousVelocity.x, m_totVelX );
  copyToDevice( m_cpuPreviousVelocity.y, m_previousVelocity.y, m_totVelY );

  hipError_t err = hipGetLastError();
  if ( err != hipSuccess ) printf( "add source copy to device: %s\n", hipGetErrorString(err) );

  for (auto &i : streams)
  {
    hipStreamCreate( &i );
  }

  d_addDensity<<<densityBlocks, threads, 0, streams[0]>>>( m_previousDensity, m_density );
  d_addVelocity_x<<<xVelocityBlocks, threads, 0, streams[1]>>>( m_previousVelocity.x, m_velocity.x );
  d_addVelocity_y<<<yVelocityBlocks, threads, 0, streams[2]>>>( m_previousVelocity.y, m_velocity.y );

  err = hipGetLastError();
  if ( err != hipSuccess ) printf( "add source kernel launch Error: %s\n", hipGetErrorString(err) );

//    int threads = 1024;
  unsigned int blocks = std::max( m_gridSize.x, m_gridSize.y ) / threads + 1;
  unsigned int blocksVelocityX = std::max( m_columnVelocity.x, m_rowVelocity.x ) / threads + 1;
  unsigned int blocksVelocityY = std::max( m_columnVelocity.y, m_rowVelocity.y ) / threads + 1;

  d_setCellBoundary<<< blocks, threads>>>( m_density );
  d_setVelBoundaryX<<< blocksVelocityX, threads, 0, streams[1]>>>( m_velocity.x );
  d_setVelBoundaryY<<< blocksVelocityY, threads, 0, streams[2]>>>( m_velocity.y );
  hipDeviceSynchronize();

  err = hipGetLastError();
  if ( err != hipSuccess ) printf( "add source Error: %s\n", hipGetErrorString(err) );
}

//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::setVel0(int i, int j, real _vx0, real _vy0)
{
  m_cpuPreviousVelocity.x[vxIdx(i, j)] = _vx0;
  m_cpuPreviousVelocity.x[vxIdx(i+1, j)] = _vx0;
  m_cpuPreviousVelocity.y[vyIdx(i, j)] = _vy0;
  m_cpuPreviousVelocity.y[vyIdx(i, j+1)] = _vy0;
}

//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::setD0(int i, int j )
{
  m_cpuPrevDensity[cIdx(i, j)] = m_inputDensity;
}


