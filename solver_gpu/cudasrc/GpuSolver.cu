#include "hip/hip_runtime.h"
#include "GpuSolverKernels.cuh"
#include "GpuSolver.h"
#include <stdio.h>
#include <iostream>
#include <fstream> 
#include <sys/time.h>
#include <time.h>

//----------------------------------------------------------------------------------------------------------------------

GpuSolver::~GpuSolver()
{
  hipFree( m_pvx );
  hipFree( m_pvy );
  hipFree( m_density );
  hipFree( m_pressure );
  hipFree( m_divergence );
  hipFree( m_velocity.x );
  hipFree( m_velocity.y );
  hipFree( m_previousVelocity.x );
  hipFree( m_previousVelocity.y );
  hipFree( m_previousDensity );
}

//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::setParameters()
{
  int mul = 1;
  m_gridSize.x = 128 * mul;
  m_gridSize.y = 128 * mul;

  m_totCell = m_gridSize.x * m_gridSize.y;
  m_rowVelocity.x = m_gridSize.x + 1;
  m_rowVelocity.y = m_gridSize.x;

  m_columnVelocity.x = m_gridSize.y;
  m_columnVelocity.y = m_gridSize.y + 1;

  m_totVelX = m_rowVelocity.x * m_columnVelocity.x;
  m_totVelY = m_rowVelocity.y * m_columnVelocity.y;

  m_min.x = 0.0f;
  m_max.x = (float)m_gridSize.x;
  m_min.y = 0.0f;
  m_max.y = (float)m_gridSize.y;

  m_timeStep = 1.0f;
  m_diffusion = 0.0f;
  m_viscosity = 0.0f;
}

//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::allocateArrays()
{
  hipMalloc( &m_pvx, sizeof(tuple<float>) * m_totVelX );
  hipMalloc( &m_pvy, sizeof(tuple<float>) * m_totVelY );
  hipMalloc( &m_density, sizeof(float)*m_totCell );
  hipMalloc( &m_pressure, sizeof(float)*m_totCell );
  hipMalloc( &m_divergence, sizeof(float)*m_totCell );
  hipMalloc( &m_velocity.x, sizeof(float)*m_totVelX );
  hipMalloc( &m_velocity.y, sizeof(float)*m_totVelY );
  hipMalloc( &m_previousVelocity.x, sizeof(float)*m_totVelX );
  hipMalloc( &m_previousVelocity.y, sizeof(float)*m_totVelY );
  hipMalloc( &m_previousDensity, sizeof(float)*m_totCell );
}

//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::activate()
{
  setParameters();
  allocateArrays();

  // 1024 -> max threads per block, in this case it will fire 16 blocks
  int nBlocks = m_totVelX / 1024;
  int blockDim = 1024 / m_gridSize.x + 1; // 9 threads per block

  dim3 block(blockDim, blockDim); // block of (X,Y) threads
  dim3 grid(nBlocks, nBlocks); // grid 2x2 blocks

  d_setPvx<<<grid, block>>>( m_pvx, m_rowVelocity.x );
  d_setPvy<<<grid, block>>>( m_pvy, m_rowVelocity.y );

//  hipDeviceSynchronize();

  hipError_t err = hipGetLastError();
  if ( err != hipSuccess ) printf("Error: %s\n", hipGetErrorString(err));
}

//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::reset()
{
  int threads = 1024;
  unsigned int densityBlocks = m_totCell / threads + 1;
  unsigned int xVelocityBlocks = m_totVelX / threads + 1;
  unsigned int yVelocityBlocks = m_totVelY / threads + 1;

  d_reset<<<densityBlocks, threads>>>(m_density, m_totCell);
  d_reset<<<xVelocityBlocks, threads>>>(m_velocity.x, m_totVelX);
  d_reset<<<yVelocityBlocks, threads>>>(m_velocity.y, m_totVelY);
//  hipDeviceSynchronize();
}

//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::cleanBuffer()
{
  int threads = 1024;
  unsigned int densityBlocks = m_totCell / threads + 1;
  unsigned int xVelocityBlocks = m_totVelX / threads + 1;
  unsigned int yVelocityBlocks = m_totVelY / threads + 1;

  d_reset<<<densityBlocks, threads>>>(m_previousDensity, m_totCell);
  d_reset<<<xVelocityBlocks, threads>>>(m_previousVelocity.x, m_totVelX);
  d_reset<<<yVelocityBlocks, threads>>>(m_previousVelocity.y, m_totVelY);
}
//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::setVelBoundary( int flag )
{
  if(flag == 1)
  {
    int threads = 1024;
    unsigned int blocks = std::max( m_columnVelocity.x, m_rowVelocity.x ) / threads + 1;
    tuple<unsigned int> size;
    size.x = m_rowVelocity.x;
    size.y = m_columnVelocity.x;
    d_setVelBoundaryX<<< blocks, threads>>>( m_velocity.x, size );
  }

  else if(flag == 2)
  {
    int threads = 1024;
    unsigned int blocks = std::max( m_columnVelocity.y, m_rowVelocity.y ) / threads + 1;
    tuple<unsigned int> size;
    size.x = m_rowVelocity.y;
    size.y = m_columnVelocity.y;
    d_setVelBoundaryY<<< blocks, threads>>>( m_velocity.y, size );
  }
}

//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::setCellBoundary(float * _value, tuple<unsigned int> _size )
{
  int threads = 1024;
  unsigned int blocks = std::max( _size.x, _size.y ) / threads + 1;

  d_setCellBoundary<<< blocks, threads>>>( _value, _size );
}

//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::projection()
{
  unsigned int bins = 81 * sizeof(float);
  int nBlocks = m_totCell / 1024;
  int blockDim = 1024 / m_gridSize.x + 1; // 9 threads per block

  dim3 block(blockDim, blockDim); // block of (X,Y) threads
  dim3 grid(nBlocks, nBlocks); // grid 2x2 blocks

  d_projection<<<grid, block, bins>>>( m_pressure, m_divergence, m_velocity, m_rowVelocity, m_columnVelocity, m_gridSize );

  hipError_t err = hipGetLastError();
  if ( err != hipSuccess ) printf("Projection Error: %s\n", hipGetErrorString(err));
}

//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::exportCSV( std::string _file, tuple<float> * _t, int _sizeX, int _sizeY )
{
  std::ofstream out;
  out.open( _file );
  out.clear();
  int totSize = _sizeX * _sizeY;
  tuple<float> * result = (tuple<float> *) malloc( sizeof( tuple<float> ) * totSize );
  if( hipMemcpy( result, _t, totSize * sizeof(tuple<float>), hipMemcpyDeviceToHost) != hipSuccess )
    exit(0);

  for(int i = 0; i < _sizeX; ++i)
  {
    for(int j = 0; j < _sizeY; ++j)
    {
      int idx = j * _sizeX + i;
      out << "( " << result[idx].x << ", " << result[idx].y << " )" << "; ";
    }
    out << "\n";
  }
  free( result );
}

//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::copy( tuple<float> * _src, tuple<float> * _dst, int _size )
{
  if( hipMemcpy( _dst, _src, _size * sizeof(tuple<float>), hipMemcpyDeviceToHost) != hipSuccess )
    exit(0);
}

//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::copy( float * _src, float * _dst, int _size )
{
  if( hipMemcpy( _dst, _src, _size * sizeof( float ), hipMemcpyDeviceToHost) != hipSuccess )
    exit(0);
}

//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::gather( float * _value, unsigned int _size )
{
  int threads = 1024;
  unsigned int blocks = _size / threads + 1;

  float * d_values;
  hipMalloc( &d_values, sizeof(float) * _size );
  if( hipMemcpy( d_values, _value, _size * sizeof( float ), hipMemcpyHostToDevice) != hipSuccess )
    exit(0);

  unsigned int bins = 10;
  d_gather<<< blocks, threads, bins * sizeof(float)>>>( d_values, _size );
  hipDeviceSynchronize();

  copy( d_values, _value, _size );

}

//----------------------------------------------------------------------------------------------------------------------


void GpuSolver::gather2D( float * _value, unsigned int _size )
{
  unsigned int bins = 10;
  // 1024 -> max threads per block, in this case it will fire 16 blocks
  int nBlocks = m_totCell / 1024;
  int blockDim = bins + 1; // 9 threads per block

  dim3 block(blockDim, blockDim); // block of (X,Y) threads
  dim3 grid(nBlocks, nBlocks); // grid 2x2 blocks

  float * d_values;
  hipMalloc( &d_values, sizeof(float) * _size );
  if( hipMemcpy( d_values, _value, _size * sizeof( float ), hipMemcpyHostToDevice) != hipSuccess )
    exit(0);

  d_gather2D<<< grid, block, bins * sizeof(float)>>>( d_values, _size );
  hipDeviceSynchronize();

  copy( d_values, _value, _size );

  hipError_t err = hipGetLastError();
  if ( err != hipSuccess ) printf(" Gather2D Error: %s\n", hipGetErrorString(err));

}

//----------------------------------------------------------------------------------------------------------------------
