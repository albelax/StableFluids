#include "hip/hip_runtime.h"
#include "GpuSolver.cuh"
#include "GpuSolver.h"
#include <stdio.h>
#include <time.h>
#include <iostream>


//----------------------------------------------------------------------------------------------------------------------

GpuSolver::GpuSolver()
{
    init();
}


//----------------------------------------------------------------------------------------------------------------------

void GpuSolver::init()
{
  m_gridSize.x = 128;
  m_gridSize.y = 128;

  m_totCell = m_gridSize.x * m_gridSize.y;
  m_rowVelocity.x = m_gridSize.x + 1;
  m_rowVelocity.y = m_gridSize.x;

  m_columnVelocity.x = m_gridSize.y;
  m_columnVelocity.y = m_gridSize.y + 1;

  m_totVelX = m_rowVelocity.x * m_columnVelocity.x;
  m_totVelY = m_rowVelocity.y * m_columnVelocity.y;

  m_min.x = 0.0f;
  m_max.x = (float)m_gridSize.x;
  m_min.y = 0.0f;
  m_max.y = (float)m_gridSize.y;

  //params
  m_timeStep = 1.0f;
  m_diffusion = 0.0f;
  m_viscosity = 0.0f;

//  hipMalloc( &m_density, sizeof(float)*m_totCell );
//  hipMalloc( &m_pressure, sizeof(float)*m_totCell );
//  hipMalloc( &m_divergence, sizeof(float)*m_totCell );
//  hipMalloc( &m_velocity.x, sizeof(float)*m_totVelX );
//  hipMalloc( &m_velocity.y, sizeof(float)*m_totVelY );
//  hipMalloc( &m_previousVelocity.x, sizeof(float)*m_totVelX );
//  hipMalloc( &m_previousVelocity.y, sizeof(float)*m_totVelY );
//  hipMalloc( &m_previousDensity, sizeof(float)*m_totCell );
  //  hipMalloc( &m_pvy, sizeof(vec2<float>)*m_totVelY );

  unsigned int N = 5000;
  unsigned int maxThreadsPerBlock = 1024;
  unsigned int numBlocks = m_totVelX / maxThreadsPerBlock + 1;

  hipMalloc( &m_pvx, sizeof(tuple<float>)*m_totVelX );
  setPvx<<<numBlocks, maxThreadsPerBlock>>>( m_pvx, m_rowVelocity.x );
  tuple<float> * result =(tuple<float> *)malloc(sizeof(tuple<float>)*m_totVelX);
  if( hipMemcpy(result, m_pvx, m_totVelX * sizeof(tuple<float>), hipMemcpyDeviceToHost) != hipSuccess)
      exit(0);

  for(int i=0; i<m_rowVelocity.x; ++i)
  {
    for(int j=0; j<m_columnVelocity.x; ++j)
    {
      std::cout << "( " << result[vxIdx(i, j)].x << "," << result[vxIdx(i, j)].y << " )  ";
    }
    std::cout << " \n";
  }
  free( result );
  hipFree( m_pvx );
}

//----------------------------------------------------------------------------------------------------------------------
// KERNELS -------------------------------------------------------------------------------------------------------------
//----------------------------------------------------------------------------------------------------------------------

// pvx[i,j] = pvx[j * size +i]
// pvy[i,j] = pvy[j * size +i]
__global__ void setPvx( tuple<float> * _pvx, unsigned int _size )
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int j = idx % _size;
    _pvx[idx].x = (float) idx;
    _pvx[idx].y = (float) j + 0.5f;
}

//----------------------------------------------------------------------------------------------------------------------

__global__ void setPvy( tuple<float> * _pvy, unsigned int _size )
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int j = idx % _size;
    _pvy[idx].x = (float) idx + 0.5f;
    _pvy[idx].y = (float) j;
}

//----------------------------------------------------------------------------------------------------------------------

__global__ void vectorAdd( float *sum, float *A, float *B, size_t arrayLength )
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if ( idx < arrayLength )
    {
        sum[idx] = A[idx] + B[idx];
    }
}

//----------------------------------------------------------------------------------------------------------------------

