#include "hip/hip_runtime.h"
#include "GpuSolverKernels.cuh"
#include "GpuSolver.h"
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <sys/time.h>
#include <time.h>


//----------------------------------------------------------------------------------------------------------------------
// KERNELS -------------------------------------------------------------------------------------------------------------
//----------------------------------------------------------------------------------------------------------------------

__global__ void d_setPvx( tuple<real> * _pvx, unsigned int _size )
{
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  int idy = threadIdx.y + blockDim.y * blockIdx.y;
  if ( idx < _size )
  {
    int i = idy * _size + idx;
    _pvx[i].x = idx;
    _pvx[i].y = idy + 0.5f;
  }
}

//----------------------------------------------------------------------------------------------------------------------

__global__ void d_setPvy( tuple<real> * _pvy, unsigned int _size )
{
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  int idy = threadIdx.y + blockDim.y * blockIdx.y;
  if ( idx < _size )
  {
    int i = idy * _size + idx;
    _pvy[i].x = (real) idx + 0.5f;
    _pvy[i].y = idy;
  }
}

//----------------------------------------------------------------------------------------------------------------------

__global__ void d_vectorAdd( real *sum, real *A, real *B, size_t arrayLength )
{
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if ( idx < arrayLength )
  {
    sum[idx] = A[idx] + B[idx];
  }
}

//----------------------------------------------------------------------------------------------------------------------

__global__ void d_reset( real * _in, unsigned int arrayLength )
{
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if ( idx < arrayLength )
  {
    _in[idx] = 0.0f;
  }
}

//----------------------------------------------------------------------------------------------------------------------

__global__ void d_setVelBoundaryX( real * _velocity, tuple<unsigned int> _size )
{
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  if ( idx > 0 && idx < _size.x - 1 ) // rowsize
  {
    _velocity[idx] =  _velocity[idx + _size.x]; // set the top row to be the same as the second row
    _velocity[idx + _size.x * (_size.y-1)] = _velocity[idx + _size.x * (_size.y - 2)]; // set the last row to be the same as the second to last row
  }

  if ( idx > 0 && idx < _size.y - 1 ) // colsize
  {
    _velocity[idx * _size.x] = -_velocity[idx * _size.x + 1]; // set the first column on the left to be the same as the next
    _velocity[idx * _size.x + ( _size.x - 1)] = -_velocity[idx * _size.x + (_size.x - 2)]; // set the first column on the right to be the same as the previous
  }

  __syncthreads();

  if ( idx == 0 )
  {
    // calculating the corners
    // horrible, wasteful way of doing it
    // but for now I just need this to work

    _velocity[0] = ( _velocity[1] + _velocity[_size.x] ) / 2;

    int dst = _size.x - 1;
    int left = _size.x - 2;
    int down = _size.x + _size.x - 1;
    _velocity[dst] = (_velocity[left] + _velocity[down])/2;

    int up = (_size.y - 1) * _size.x + 1;
    left = (_size.y - 2) * _size.x;
    dst = (_size.y - 1) * _size.x;
    _velocity[dst] = (_velocity[up] + _velocity[left])/2;

    dst = (_size.y - 1) * _size.x + (_size.x -1);
    left = (_size.y - 1) * _size.x + (_size.x - 2);
    up = (_size.y - 2) * _size.x + (_size.x - 1);

    _velocity[dst] = ( _velocity[left] + _velocity[up] ) / 2;
  }
  __syncthreads();

}

//----------------------------------------------------------------------------------------------------------------------

__global__ void d_setVelBoundaryY( real * _velocity, tuple<unsigned int> _size )
{
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  if ( idx > 0 && idx < _size.x - 1 ) // rowsize
  {
    _velocity[idx] = - _velocity[idx + _size.x]; // set the top row to be the same as the second row
    _velocity[idx + _size.x * (_size.y-1)] = -_velocity[idx + _size.x * (_size.y - 2)]; // set the last row to be the same as the second to last row

  }

  if ( idx > 0 && idx < _size.y - 1 ) // colsize
  {
    _velocity[idx * _size.x] = _velocity[idx * _size.x + 1]; // set the first column on the left to be the same as the next
    _velocity[idx * _size.x + ( _size.x - 1)] = _velocity[idx * _size.x + (_size.x - 2)]; // set the first column on the right to be the same as the previous

  }

  __syncthreads();

  if ( idx == 0 )
  {
    // calculating the corners
    // horrible, wasteful way of doing it
    // but for now I just need this to work

    _velocity[0] = ( _velocity[1] + _velocity[_size.x] ) / 2;

    int dst = _size.x - 1;
    int left = _size.x - 2;
    int down = _size.x + _size.x - 1;
    _velocity[dst] = (_velocity[left] + _velocity[down])/2;

    int up = (_size.y - 1) * _size.x + 1;
    left = (_size.y - 2) * _size.x;
    dst = (_size.y - 1) * _size.x;
    _velocity[dst] = (_velocity[up] + _velocity[left])/2;

    dst = (_size.y - 1) * _size.x + (_size.x -1);
    left = (_size.y - 1) * _size.x + (_size.x - 2);
    up = (_size.y - 2) * _size.x + (_size.x - 1);

    _velocity[dst] = ( _velocity[left] + _velocity[up] ) / 2;
  }
}

//----------------------------------------------------------------------------------------------------------------------

__global__ void d_setCellBoundary( real * _value , tuple<unsigned int> _size )
{
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  if ( idx > 0 && idx < _size.x - 1 )
  {
    _value[idx] = _value[idx + _size.x];
    _value[idx + _size.x * (_size.y - 1)] = _value[idx + _size.x * (_size.y - 2)];
  }

  if ( idx > 0 && idx < _size.y - 1 )
  {
    _value[idx * _size.x] = _value[idx * _size.x + 1]; // set the first column on the left to be the same as the next
    _value[idx * _size.x + ( _size.x - 1)] = _value[idx * _size.x + (_size.x - 2)];
  }

  if ( idx == 0 )
  {
    // again
    // calculating the corners
    // horrible, wasteful way of doing it
    // but for now I just need this to work

    _value[0] = ( _value[1] + _value[_size.x] ) / 2;

    int dst = _size.x - 1;
    int left = _size.x - 2;
    int down = _size.x + _size.x - 1;
    _value[dst] = (_value[left] + _value[down]) / 2;

    int up = (_size.y - 1) * _size.x + 1;
    left = (_size.y - 2) * _size.x;
    dst = (_size.y - 1) * _size.x;
    _value[dst] = (_value[up] + _value[left])/2;

    dst = (_size.y - 1) * _size.x + (_size.x -1);
    left = (_size.y - 1) * _size.x + (_size.x - 2);
    up = (_size.y - 2) * _size.x + (_size.x - 1);

    _value[dst] = ( _value[left] + _value[up] ) / 2;
  }
}

//----------------------------------------------------------------------------------------------------------------------

__global__ void d_gather( real * _value, unsigned int _size )
{
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  extern __shared__ real localValue[];

  if ( idx > 0 && idx < _size - 1 )
  {
    localValue[idx] = ( _value[idx - 1] + _value[idx] + _value[idx + 1] );
    __syncthreads();
    _value[idx] = localValue[idx];
  }
}

//----------------------------------------------------------------------------------------------------------------------

__global__ void d_projection( real * _pressure, real * _divergence, tuple<unsigned int> _gridSize)
{
  // projection Step
  // this should be in a loop...
  extern __shared__ real local_pressure[];

  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  int idy = threadIdx.y + blockDim.y * blockIdx.y;

  if ( idx > 0 && idx < _gridSize.x - 1 &&
       idy > 0 && idy < _gridSize.y - 1 )
  {

    int sIdx = threadIdx.y * blockDim.x + threadIdx.x;
    int currentCell = idy * _gridSize.x + idx;

    int right = idy * _gridSize.x + (idx + 1);
    int left = idy * _gridSize.x + (idx - 1);
    int down = (idy + 1) * _gridSize.x + idx;
    int up = (idy - 1) * _gridSize.x + idx;

    local_pressure[sIdx] = ( _pressure[right] + _pressure[left] + _pressure[down] + _pressure[up] - _divergence[currentCell])/4.0;
    //    __syncthreads();

    _pressure[currentCell] = local_pressure[sIdx];
    //    __syncthreads();
  }
}

//----------------------------------------------------------------------------------------------------------------------

__global__ void d_divergenceStep(real * _pressure, real * _divergence, tuple<real *> _velocity,
                                 tuple<unsigned int> _rowVelocity, tuple<unsigned int> _gridSize)
{
  // memory shared within the block, I will treat this as a tiny 2D array,
  // the size is decided outside the kernel,
  // if the # of threads in a block is 9 the size will be 81 ( array[9][9] )
  extern __shared__ real local_divergence[];

  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  int idy = threadIdx.y + blockDim.y * blockIdx.y;

  if ( idx > 0 && idx < _gridSize.x - 1 &&
       idy > 0 && idy < _gridSize.y - 1 )
  {
    int sIdx = threadIdx.y * blockDim.x + threadIdx.x;

    int currentCell = idy * _gridSize.x + idx;
    int right = idy * _rowVelocity.x + (idx + 1);
    int currentVelX = idy * _rowVelocity.x + idx;
    int down = (idy + 1) * _rowVelocity.y + idx;
    int currentVelY = idy * _rowVelocity.y + idx;

    // index of the shared memory
    local_divergence[sIdx] = _velocity.x[right] - _velocity.x[currentVelX] + _velocity.y[down] - _velocity.y[currentVelY];

    _pressure[currentCell] = 0.0;
    _divergence[currentCell] = local_divergence[sIdx];
    //    __syncthreads();
  }
}

//----------------------------------------------------------------------------------------------------------------------

__global__ void d_velocityStep(real * _pressure, tuple<real *> _velocity,
                               tuple<unsigned int> _rowVelocity, tuple<unsigned int> _columnVelocity,
                               tuple<unsigned int> _gridSize)
{
  extern __shared__ real local_velocity[];

  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  int idy = threadIdx.y + blockDim.y * blockIdx.y;

  if ( idx > 0 && idx < _rowVelocity.x - 1 &&
       idy > 0 && idy < _columnVelocity.x - 1 )
  {
    int velocityIdx = idy * _rowVelocity.x + idx;
    int sIdx = threadIdx.y * blockDim.x + threadIdx.x;

    int cellIdx = idy * _gridSize.x + idx;
    int cellLeft = idy * _gridSize.x + (idx - 1);

    local_velocity[sIdx] = _pressure[cellIdx] - _pressure[cellLeft];
    //    __syncthreads();
    _velocity.x[velocityIdx] -= local_velocity[sIdx];
  }

  if ( idx > 0 && idx < _rowVelocity.y - 1 &&
       idy > 0 && idy < _columnVelocity.y - 1 )
  {

    int velocityIdx = idy * _rowVelocity.y + idx;
    int sIdx = threadIdx.y * blockDim.x + threadIdx.x;

    int cellIdx = idy * _gridSize.x + idx;
    int cellUp = (idy-1) * _gridSize.x + idx;

    local_velocity[sIdx] = _pressure[cellIdx] - _pressure[cellUp];
    _velocity.y[velocityIdx] -= local_velocity[sIdx];
  }
}


//----------------------------------------------------------------------------------------------------------------------
//int vxIdx(int i, int j){ return j*m_rowVelocity.x+i; }
//int vyIdx(int i, int j){ return j*m_rowVelocity.y+i; }
//int cIdx(int i, int j){ return j*m_gridSize.x+i; }

__global__ void d_advectVelocity( tuple<real *> _previousVelocity, tuple<real *> _velocity,
                                  tuple<real> * _pvx, tuple<real> * _pvy,
                                  tuple<unsigned int> _rowVelocity,
                                  tuple<unsigned int> _columnVelocity,
                                  tuple<unsigned int> _gridSize )
{
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  int idy = threadIdx.y + blockDim.y * blockIdx.y;
  unsigned short currentIdx = idy * _rowVelocity.x + idx;
  unsigned short currentIdy = idy * _rowVelocity.y + idx;

  if ( idx > 0 && idx < _rowVelocity.x - 1 &&
       idy > 0 && idy < _columnVelocity.x - 1 )
  {
    real nvx = _previousVelocity.x[currentIdx];
    real nvy = (_previousVelocity.y[idy * _rowVelocity.y + idx-1] +
        _previousVelocity.y[(idy + 1) * _rowVelocity.y + (idx - 1)] +
        _previousVelocity.y[currentIdy]+
        _previousVelocity.y[(idy + 1) * _rowVelocity.y + idx])/4;

    real oldX = _pvx[currentIdx].x - nvx * 1;
    real oldY = _pvx[currentIdx].y - nvy * 1;

    if(oldX < 0.5f) oldX = 0.5f;
    if(oldX > _gridSize.x-0.5f) oldX = _gridSize.x-0.5f;
    if(oldY < 1.0f) oldY = 1.0f;
    if(oldY > _gridSize.y-1.0f) oldY = _gridSize.y-1.0f;

    int i0 = (int)oldX;
    int j0 = (int)(oldY-0.5f);
    int i1 = i0+1;
    int j1 = j0+1;

    real wL = _pvx[j0 * _rowVelocity.x + i1].x-oldX;
    real wR = 1.0f-wL;
    real wB = _pvx[j1 * _rowVelocity.x + i0].y-oldY;
    real wT = 1.0f-wB;

    _velocity.x[currentIdx] = wB * (wL * _previousVelocity.x[j0 * _rowVelocity.x + i0] +
        wR * _previousVelocity.x[j0 * _rowVelocity.x + i1]) +
        wT * (wL * _previousVelocity.x[j1 * _rowVelocity.x + i0] +
        wR * _previousVelocity.x[j1 * _rowVelocity.x + i1]);
  }

  if ( idx > 0 && idx < _rowVelocity.y - 1 &&
       idy > 0 && idy < _columnVelocity.y - 1 )
  {
    real nvx = (
        _previousVelocity.x[(idy - 1) * _rowVelocity.x + idx]+
        _previousVelocity.x[(idy - 1) * _rowVelocity.x + (idx + 1)] +
        _previousVelocity.x[currentIdx]+
        _previousVelocity.x[idy * _rowVelocity.x + (idx + 1)]
        )/4;

    real nvy = _previousVelocity.y[currentIdy];

    real oldX = _pvy[currentIdy].x - nvx*1;
    real oldY = _pvy[currentIdy].y - nvy*1;

    if(oldX < 1.0f) oldX = 1.0f;
    if(oldX > _gridSize.x-1.0f) oldX = _gridSize.x-1.0f;
    if(oldY < 0.5f) oldY = 0.5f;
    if(oldY > _gridSize.y-0.5f) oldY = _gridSize.y-0.5f;

    int i0 = (int)(oldX-0.5f);
    int j0 = (int)oldY;
    int i1 = i0+1;
    int j1 = j0+1;

    real wL = _pvy[j0 * _rowVelocity.y + i1].x-oldX;
    real wR = 1.0f-wL;
    real wB = _pvy[j1 * _rowVelocity.y + i0].y-oldY;
    real wT = 1.0f-wB;

    _velocity.y[currentIdy] = wB * (wL * _previousVelocity.y[j0 * _rowVelocity.y + i0] +
        wR * _previousVelocity.y[j0 * _rowVelocity.y + i1]) +
        wT * (wL * _previousVelocity.y[j1 * _rowVelocity.y + i0] +
        wR * _previousVelocity.y[j1 * _rowVelocity.y + i1]);
//    _velocity.y[currentIdy] = nvx;
  }
}

//----------------------------------------------------------------------------------------------------------------------

