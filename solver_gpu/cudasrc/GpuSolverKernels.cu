#include "hip/hip_runtime.h"
#include "GpuSolverKernels.cuh"
#include "GpuSolver.h"
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <sys/time.h>
#include <time.h>


//----------------------------------------------------------------------------------------------------------------------
// KERNELS -------------------------------------------------------------------------------------------------------------
//----------------------------------------------------------------------------------------------------------------------

__global__ void d_setPvx( tuple<float> * _pvx, unsigned int _size )
{
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  int idy = threadIdx.y + blockDim.y * blockIdx.y;
  if ( idx < _size )
  {
    int i = idy * _size + idx;
    _pvx[i].x = idx;
    _pvx[i].y = idy + 0.5f;
  }
}

//----------------------------------------------------------------------------------------------------------------------

__global__ void d_setPvy( tuple<float> * _pvy, unsigned int _size )
{
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  int idy = threadIdx.y + blockDim.y * blockIdx.y;
  if ( idx < _size )
  {
    int i = idy * _size + idx;
    _pvy[i].x = (float) idx + 0.5f;
    _pvy[i].y = idy;
  }
}

//----------------------------------------------------------------------------------------------------------------------

__global__ void d_vectorAdd( float *sum, float *A, float *B, size_t arrayLength )
{
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if ( idx < arrayLength )
  {
    sum[idx] = A[idx] + B[idx];
  }
}

//----------------------------------------------------------------------------------------------------------------------

__global__ void d_reset( float * _in, unsigned int arrayLength )
{
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if ( idx < arrayLength )
  {
    _in[idx] = 0.0f;
  }
}

//----------------------------------------------------------------------------------------------------------------------

__global__ void d_setVelBoundaryX( float * _velocity, tuple<unsigned int> _size )
{
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  if ( idx > 0 && idx < _size.x - 1 ) // rowsize
  {
    _velocity[idx] =  _velocity[idx + _size.x]; // set the top row to be the same as the second row
    _velocity[idx + _size.x * (_size.y-1)] = _velocity[idx + _size.x * (_size.y - 2)]; // set the last row to be the same as the second to last row
  }

  if ( idx > 0 && idx < _size.y - 1 ) // colsize
  {
    _velocity[idx * _size.x] = -_velocity[idx * _size.x + 1]; // set the first column on the left to be the same as the next
    _velocity[idx * _size.x + ( _size.x - 1)] = -_velocity[idx * _size.x + (_size.x - 2)]; // set the first column on the right to be the same as the previous
  }

  __syncthreads();

  if ( idx == 0 )
  {
    // calculating the corners
    // horrible, wasteful way of doing it
    // but for now I just need this to work

    _velocity[0] = ( _velocity[1] + _velocity[_size.x] ) / 2;

    int dst = _size.x - 1;
    int left = _size.x - 2;
    int down = _size.x + _size.x - 1;
    _velocity[dst] = (_velocity[left] + _velocity[down])/2;

    int up = (_size.y - 1) * _size.x + 1;
    left = (_size.y - 2) * _size.x;
    dst = (_size.y - 1) * _size.x;
    _velocity[dst] = (_velocity[up] + _velocity[left])/2;

    dst = (_size.y - 1) * _size.x + (_size.x -1);
    left = (_size.y - 1) * _size.x + (_size.x - 2);
    up = (_size.y - 2) * _size.x + (_size.x - 1);

    _velocity[dst] = ( _velocity[left] + _velocity[up] ) / 2;
  }
}

//----------------------------------------------------------------------------------------------------------------------

__global__ void d_setVelBoundaryY( float * _velocity, tuple<unsigned int> _size )
{
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  if ( idx > 0 && idx < _size.x - 1 ) // rowsize
  {
    _velocity[idx] = - _velocity[idx + _size.x]; // set the top row to be the same as the second row
    _velocity[idx + _size.x * (_size.y-1)] = -_velocity[idx + _size.x * (_size.y - 2)]; // set the last row to be the same as the second to last row

  }

  if ( idx > 0 && idx < _size.y - 1 ) // colsize
  {
    _velocity[idx * _size.x] = _velocity[idx * _size.x + 1]; // set the first column on the left to be the same as the next
    _velocity[idx * _size.x + ( _size.x - 1)] = _velocity[idx * _size.x + (_size.x - 2)]; // set the first column on the right to be the same as the previous

  }

  __syncthreads();

  if ( idx == 0 )
  {
    // calculating the corners
    // horrible, wasteful way of doing it
    // but for now I just need this to work

    _velocity[0] = ( _velocity[1] + _velocity[_size.x] ) / 2;

    int dst = _size.x - 1;
    int left = _size.x - 2;
    int down = _size.x + _size.x - 1;
    _velocity[dst] = (_velocity[left] + _velocity[down])/2;

    int up = (_size.y - 1) * _size.x + 1;
    left = (_size.y - 2) * _size.x;
    dst = (_size.y - 1) * _size.x;
    _velocity[dst] = (_velocity[up] + _velocity[left])/2;

    dst = (_size.y - 1) * _size.x + (_size.x -1);
    left = (_size.y - 1) * _size.x + (_size.x - 2);
    up = (_size.y - 2) * _size.x + (_size.x - 1);

    _velocity[dst] = ( _velocity[left] + _velocity[up] ) / 2;
  }
}

//----------------------------------------------------------------------------------------------------------------------

__global__ void d_setCellBoundary( float * _value , tuple<unsigned int> _size )
{
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  if ( idx > 0 && idx < _size.x - 1 )
  {
    _value[idx] = _value[idx + _size.x];
    _value[idx + _size.x * (_size.y-1)] = _value[idx + _size.x * (_size.y - 2)];
  }

  if ( idx > 0 && idx < _size.y - 1 )
  {
    _value[idx * _size.x] = _value[idx * _size.x + 1]; // set the first column on the left to be the same as the next
    _value[idx * _size.x + ( _size.x - 1)] = _value[idx * _size.x + (_size.x - 2)];
  }
  __syncthreads();

  if ( idx == 0 )
  {
    // again
    // calculating the corners
    // horrible, wasteful way of doing it
    // but for now I just need this to work

    _value[0] = ( _value[1] + _value[_size.x] ) / 2;

    int dst = _size.x - 1;
    int left = _size.x - 2;
    int down = _size.x + _size.x - 1;
    _value[dst] = (_value[left] + _value[down]) / 2;

    int up = (_size.y - 1) * _size.x + 1;
    left = (_size.y - 2) * _size.x;
    dst = (_size.y - 1) * _size.x;
    _value[dst] = (_value[up] + _value[left])/2;

    dst = (_size.y - 1) * _size.x + (_size.x -1);
    left = (_size.y - 1) * _size.x + (_size.x - 2);
    up = (_size.y - 2) * _size.x + (_size.x - 1);

    _value[dst] = ( _value[left] + _value[up] ) / 2;
  }

}

//----------------------------------------------------------------------------------------------------------------------

__global__ void d_gather( float * _value, unsigned int _size )
{
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  extern __shared__ float localValue[];

  if ( idx > 0 && idx < _size - 1 )
  {
    //    printf(" idx: %d, idy: %d\n", idx, idy );
    localValue[idx] = ( _value[idx - 1] + _value[idx] + _value[idx + 1] );
    __syncthreads();
    _value[idx] = localValue[idx];

  }
}

//----------------------------------------------------------------------------------------------------------------------

__global__ void d_gather2D( float * _value, unsigned int _size )
{
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  int idy = threadIdx.y + blockDim.y * blockIdx.y;

  extern __shared__ float localValue[];

  if ( idx < _size )
  {
    //    printf(" idx: %d, idy: %d\n", idx, idy );
    localValue[idx] = idx - idy; //( _value[idx - 1] + _value[idx] + _value[idx + 1] );
    __syncthreads();
    _value[idx] = localValue[idx];
  }
}



//----------------------------------------------------------------------------------------------------------------------

__global__ void d_projection( float * _pressure, float * _divergence, tuple<float *> _velocity,
                              tuple<unsigned int> _rowVelocity, tuple<unsigned int> _columnVelocity,
                              tuple<unsigned int> _gridSize)
{
  extern __shared__ float local_divergence[];

  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  int idy = threadIdx.y + blockDim.y * blockIdx.y;

  if ( idx > 0 && idx < _gridSize.x - 1 &&
       idy > 0 && idy < _gridSize.y - 1 )
  {
    int currentCell = idy * _gridSize.x + idx;

    int right = idy * _rowVelocity.x + (idx + 1);
    int currentVelX = idy * _rowVelocity.x + idx;
    int down = (idy + 1) * _rowVelocity.y + idx;
    int currentVelY = idy * _rowVelocity.y + idx;

    int sIdx = threadIdx.y * 9 + threadIdx.x;
    local_divergence[sIdx] = _velocity.x[right] - _velocity.x[currentVelX] + _velocity.y[down] - _velocity.y[currentVelY];

    _pressure[currentCell] = 0;
    _divergence[currentCell] = local_divergence[sIdx];
  }
}

//----------------------------------------------------------------------------------------------------------------------
