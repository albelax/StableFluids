#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <stdio.h>
#include <time.h>
#include <iostream>
#include "rand_gpu.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

#define CUDA_CALL(x) {\
    if( (x) !=hipSuccess) {\
    printf("CUDA failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(x));\
    exit(0);\
    }\
    }

#define CURAND_CALL(x) {\
    if((x)!=HIPRAND_STATUS_SUCCESS) {\
    printf("CURAND failure at %s:%d\n",__FILE__,__LINE__);\
    exit(0);\
    }\
    }

/**
 * Fill an array with random floats using the CURAND function.
 * \param devData The chunk of GPU memory you want to fill with floats within the range (0,1]
 * \param n The size of the chunk of data
 */
int Rand_GPU::randFloatsInternal(float *&devData, const size_t n)
{
    // The generator, used for random numbers
    hiprandGenerator_t gen;

    // Create pseudo-random number generator
    CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    
    // Set seed to be the current time (note that calls close together will have same seed!)
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL)));

    // Generate n floats on device
    CURAND_CALL(hiprandGenerateUniform(gen, devData, n));

    // Cleanup
    CURAND_CALL(hiprandDestroyGenerator(gen));
    return EXIT_SUCCESS;
}

/**
 * This function takes an stl vector by reference and fills it up with random numbers generated on the GPU
 * \param tgt The target vector to fill
 * \return EXIT_SUCCESS if everything went well
 */
int Rand_GPU::randFloats(std::vector<float>& tgt)
{
    int ret_val = EXIT_SUCCESS;
    // Create a device array using CUDA
    float *d_Rand_ptr;
    CUDA_CALL(hipMalloc(&d_Rand_ptr, tgt.size() * sizeof(float)));

    // Fill the thrust vector using the randFloats() function
    randFloatsInternal(d_Rand_ptr, tgt.size());

    // Copy the data back to the input vector
    float *h_Rand_ptr = (float*) malloc(tgt.size() * sizeof(float));

    // Need to check if the malloc was successful
    if (h_Rand_ptr != NULL)
    {
        // Copy the memory to the local pointer
        CUDA_CALL(hipMemcpy(h_Rand_ptr, d_Rand_ptr, sizeof(float) * tgt.size(), hipMemcpyDeviceToHost));

        // Transfer this memory into the target structure
        std::copy(h_Rand_ptr, h_Rand_ptr + tgt.size(), tgt.begin());

        // Free up the local memory
        free(h_Rand_ptr);
    } else {
        // The memory allocation failed so this will ensure the exit is "graceful"
        ret_val = EXIT_FAILURE;
    }

    // Free up the gpu memory
    hipFree(d_Rand_ptr);

    // Return success
    return ret_val;
}

__global__ void cudaHello()
{
    printf("hello from thread %d in block %d \n", threadIdx.x, blockIdx.x );
}

__global__ void add(float *sum,float *A,float *B, size_t arrayLength)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < arrayLength)
    {
        sum[idx] = A[idx] + B[idx];
    }
}

void Rand_GPU::hello()
{
    int N = 5000;
    int maxThreadsPerBlock = 1024;
    int numBlocks = N / maxThreadsPerBlock + 1;
    cudaHello<<<numBlocks, maxThreadsPerBlock>>>();
}


void Pippo::print()
{
    int N = 5000;
    int maxThreadsPerBlock = 1024;
    int numBlocks = N / maxThreadsPerBlock + 1;

    printf("pippo here \n");

    float *host_A = (float*) malloc(N * sizeof(float));
    if (host_A == NULL) exit(0);

    int i;
    for (i=0; i < N; ++i)
    {
        host_A[i] = float(i);
    }
    float *dev_A, *dev_B, *dev_C;
    if (hipMalloc(&dev_A, N * sizeof(float)) != hipSuccess)
    {
        exit(0);
    }

    if (hipMalloc(&dev_B, N * sizeof(float)) != hipSuccess)
    {
        exit(0);
    }
    if (hipMalloc(&dev_C, N * sizeof(float)) != hipSuccess)
    {
        exit(0);
    }

    if (hipMemcpy(dev_A, host_A,
                   N * sizeof(float),
                   hipMemcpyHostToDevice) != hipSuccess) exit(0);
    if (hipMemcpy (dev_B, host_A,
                    N * sizeof(float),
                    hipMemcpyHostToDevice) != hipSuccess) exit(0);

    add<<<numBlocks, maxThreadsPerBlock>>>(dev_C, dev_A, dev_B, N);
    float *result =(float*) malloc(N * sizeof(float));
    if( hipMemcpy(result, dev_C, N * sizeof(float), hipMemcpyDeviceToHost) != hipSuccess) exit(0);

    hipDeviceSynchronize();
    for (int i = 0; i < N; ++i)
    {
        printf("The result of vector %d is %d \n", i, result[i]);
    }
}


