#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <stdio.h>
#include <time.h>
#include <iostream>
#include "rand_gpu.h"

#define CUDA_CALL(x) {\
    if( (x) !=hipSuccess) {\
        printf("CUDA failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(x));\
        exit(0);\
    }\
}

#define CURAND_CALL(x) {\
    if((x)!=HIPRAND_STATUS_SUCCESS) {\
        printf("CURAND failure at %s:%d\n",__FILE__,__LINE__);\
        exit(0);\
    }\
}

/**
 * Fill an array with random floats using the CURAND function.
 * \param devData The chunk of GPU memory you want to fill with floats within the range (0,1]
 * \param n The size of the chunk of data
 */
int Rand_GPU::randFloatsInternal(float *&devData, const size_t n)
{
    // The generator, used for random numbers
    hiprandGenerator_t gen;

    // Create pseudo-random number generator
    CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    
    // Set seed to be the current time (note that calls close together will have same seed!)
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL)));

    // Generate n floats on device
    CURAND_CALL(hiprandGenerateUniform(gen, devData, n));

    // Cleanup
    CURAND_CALL(hiprandDestroyGenerator(gen));
    return EXIT_SUCCESS;
}

/**
 * This function takes an stl vector by reference and fills it up with random numbers generated on the GPU
 * \param tgt The target vector to fill
 * \return EXIT_SUCCESS if everything went well
 */
int Rand_GPU::randFloats(std::vector<float>& tgt)
{
    int ret_val = EXIT_SUCCESS;
    // Create a device array using CUDA
    float *d_Rand_ptr;
    CUDA_CALL(hipMalloc(&d_Rand_ptr, tgt.size() * sizeof(float)));

    // Fill the thrust vector using the randFloats() function    
    //randSphereCoords(d_Rand_ptr, tgt.size());
    randFloatsInternal(d_Rand_ptr, tgt.size());

    // Copy the data back to the input vector
    float *h_Rand_ptr = (float*) malloc(tgt.size() * sizeof(float));

    // Need to check if the malloc was successful
    if (h_Rand_ptr != NULL) {
        // Copy the memory to the local pointer
        CUDA_CALL(hipMemcpy(h_Rand_ptr, d_Rand_ptr, sizeof(float) * tgt.size(), hipMemcpyDeviceToHost));

        // Transfer this memory into the target structure
        std::copy(h_Rand_ptr, h_Rand_ptr + tgt.size(), tgt.begin());

        // Free up the local memory
        free(h_Rand_ptr);        
    } else {
        // The memory allocation failed so this will ensure the exit is "graceful"
        ret_val = EXIT_FAILURE;
    }

    // Free up the gpu memory
    hipFree(d_Rand_ptr);        

    // Return success
    return ret_val;
}
